#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <unistd.h>

#define ALLOC_SIZE 1024000
#define hipSuccess hipSuccess 

int GetCudaFrequency() {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  return prop.clockRate;
}

__global__ void SpinForever() {
  int i = 0;
  while (i == 0) {}
}

// Spin in the GPU for a certain delay, siAAmulates load to induce synchronization delay.
__global__ void LaunchSpinForTime(uint64_t delay) {
    uint64_t start_clock = clock64();
    uint64_t clock_offset = 0;
    while (clock_offset < delay)
    {
        clock_offset = clock64() - start_clock;
    }
}


int ExperimentNoUse(int cpuDelay) {
  
  uint64_t gpuDelay = (cpuDelay + (cpuDelay * 0.20)) * (GetCudaFrequency() * 1000);
  fprintf(stderr, "GPU will delay for - %ull clock cycles (%ull seconds)\n",gpuDelay, gpuDelay / (GetCudaFrequency() * 1000) );
  //uint64_t *x, *d_x, *d_y;
  // Launch GPU Kernel
  LaunchSpinForTime<<<1,1>>>(gpuDelay);
  //LaunchSpinOnMem<<<1,1>>>(flags);
  // Immediately sleep to simulate CPU workload
  sleep(cpuDelay);
  fprintf(stderr, "Left CPU Work, waiting on GPU synchronization\n");
  // Unnnecessary Synchronization that should take between 10-20% of execution time
  hipDeviceSynchronize();
  fprintf(stderr, "GPU synchronization complete\n");
  sleep(15);
  return 0;
}

int myexperiment() {
  fprintf(stderr, "Cycle count: %d\n", GetCudaFrequency());
  SpinForever<<<1,1>>>();
  sleep(5);
  fprintf(stderr, "Cycle count: %d\n", GetCudaFrequency());
  uint64_t *d_x;
  if (hipSuccess != hipMalloc(&d_x, ALLOC_SIZE*sizeof(uint64_t)))
   fprintf(stderr, "ERROR\n"); 
  for (int i = 0; i < 2; i++)
    ExperimentNoUse(60);
  return 0;
}

int main() {
    return myexperiment();
}
